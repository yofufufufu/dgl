#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2021 by Contributors
 * @file array/cuda/rowwise_sampling.cu
 * @brief uniform rowwise sampling
 */

#include <hiprand/hiprand_kernel.h>
#include <dgl/random.h>
#include <dgl/runtime/device_api.h>

#include <stdgpu/unordered_set.cuh>
#include <stdgpu/bitset.cuh>
#include <stdgpu/queue.cuh>
#include <stdgpu/deque.cuh>
#include <stdgpu/vector.cuh>

#include <numeric>
#include <nvtx3/nvToolsExt.h>

#include "../../array/cuda/atomic.cuh"
#include "../../runtime/cuda/cuda_common.h"
#include "./dgl_cub.cuh"
#include "./utils.h"

namespace dgl {
using namespace cuda;
using namespace aten::cuda;
namespace aten {
namespace impl {

namespace {

constexpr int BLOCK_SIZE = 128;

/**
 * @brief Compute the size of each row in the sampled CSR, without replacement.
 *
 * @tparam IdType The type of node and edge indexes.
 * @param num_picks The number of non-zero entries to pick per row.
 * @param num_rows The number of rows to pick.
 * @param in_rows The set of rows to pick.
 * @param in_ptr The index where each row's edges start.
 * @param out_deg The size of each row in the sampled matrix, as indexed by
 * `in_rows` (output).
 */
template <typename IdType>
__global__ void _CSRRowWiseSampleDegreeKernel(
    const int64_t num_picks, const int64_t num_rows,
    const IdType* const in_rows, const IdType* const in_ptr,
    IdType* const out_deg) {
  const int tIdx = threadIdx.x + blockIdx.x * blockDim.x;

  if (tIdx < num_rows) {
    const int in_row = in_rows[tIdx];
    const int out_row = tIdx;
    out_deg[out_row] = min(
        static_cast<IdType>(num_picks), in_ptr[in_row + 1] - in_ptr[in_row]);

    if (out_row == num_rows - 1) {
      // make the prefixsum work
      out_deg[num_rows] = 0;
    }
  }
}

/**
 * @brief Compute the size of each row in the sampled CSR, with replacement.
 *
 * @tparam IdType The type of node and edge indexes.
 * @param num_picks The number of non-zero entries to pick per row.
 * @param num_rows The number of rows to pick.
 * @param in_rows The set of rows to pick.
 * @param in_ptr The index where each row's edges start.
 * @param out_deg The size of each row in the sampled matrix, as indexed by
 * `in_rows` (output).
 */
template <typename IdType>
__global__ void _CSRRowWiseSampleDegreeReplaceKernel(
    const int64_t num_picks, const int64_t num_rows,
    const IdType* const in_rows, const IdType* const in_ptr,
    IdType* const out_deg) {
  const int tIdx = threadIdx.x + blockIdx.x * blockDim.x;

  if (tIdx < num_rows) {
    const int64_t in_row = in_rows[tIdx];
    const int64_t out_row = tIdx;

    if (in_ptr[in_row + 1] - in_ptr[in_row] == 0) {
      out_deg[out_row] = 0;
    } else {
      out_deg[out_row] = static_cast<IdType>(num_picks);
    }

    if (out_row == num_rows - 1) {
      // make the prefixsum work
      out_deg[num_rows] = 0;
    }
  }
}

// the sample result of a hop
struct selectedEdgeInfo {
    short hop;
    int64_t row_num;
    int64_t col_num;
    int64_t data_num;
};

struct CheckHopNum {
    int hop_num;
    CheckHopNum(int num) : hop_num(num) {};
    __host__ __device__
    // num from stencil
    bool operator()(const int64_t &num) {
        return num == hop_num;
    }
};

struct RowTrans : public thrust::unary_function<selectedEdgeInfo, int64_t> {
    __host__ __device__
    int64_t operator()(const selectedEdgeInfo &tuple) {
        return tuple.row_num;
    }
};

struct ColTrans : public thrust::unary_function<selectedEdgeInfo, int64_t> {
    __host__ __device__
    int64_t operator()(const selectedEdgeInfo &tuple) {
        return tuple.col_num;
    }
};

struct DataTrans : public thrust::unary_function<selectedEdgeInfo, int64_t> {
    __host__ __device__
    int64_t operator()(const selectedEdgeInfo &tuple) {
        return tuple.data_num;
    }
};

struct HopTrans : public thrust::unary_function<selectedEdgeInfo, int> {
    __host__ __device__
    int operator()(const selectedEdgeInfo &tuple) {
        return tuple.hop;
    }
};

// should try to push the node with more edges first
__global__ void queue_init(
        stdgpu::queue<thrust::pair<short, int64_t>> queue, uint* bits, const int64_t* const in_rows,
        const int64_t num_rows, const int hops, const int64_t total_num_rows) {
    const int tIdx = threadIdx.x + blockIdx.x * blockDim.x;
//    if (tIdx < num_rows * hops) {
//        int hop_num = 1 + tIdx / num_rows;
//        queue.push({hop_num, in_rows[tIdx % num_rows]});
//        if (hop_num > 1)
//            bits[in_rows[tIdx % num_rows] + total_num_rows * (hop_num - 2)] = 1;
//    }
    if (tIdx < num_rows) {
        queue.push({1, in_rows[tIdx]});
    }
}

__launch_bounds__(128) __global__ void _CSRRowWiseSampleUniformTaskParallelismKernel(
        const uint64_t rand_seed, const int64_t * num_picks, const int64_t * const in_rows,
        const int64_t num_rows, const int hops, const int64_t total_num_rows,
        const int64_t * const in_ptr, const int64_t * const in_index, const int64_t * const data,
        stdgpu::queue<thrust::pair<short, int64_t>> task_queue,
        uint* bits,
        stdgpu::vector<selectedEdgeInfo> result
        ) {
    __shared__ int64_t blockTask[2];
    __shared__ bool sharedRes[1];
    // num_pick cannot be larger than 128
    // any better solution?
    __shared__ int64_t permList[128];

    // do not use separate init kernel maybe faster? the result seems correct although only block level sync
    // not correct! convergence will be changed(accuracy~0.6, correct accuracy~0.7)
//    const int tIdx = threadIdx.x + blockIdx.x * blockDim.x;
//    if (tIdx < num_rows) {
//        task_queue.push({1, in_rows[tIdx]});
//    }
//    __syncthreads();

    hiprandStatePhilox4_32_10_t rng;
    // different block has different seed
    // different thread in block has different (sub)sequence
    hiprand_init(rand_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);
//    hiprand_init(rand_seed, 0, 0, &rng);

    while (true) {
        if (threadIdx.x == 0) {
            auto pop_res = task_queue.pop();
            sharedRes[0] = pop_res.second;
            if (pop_res.second) {
                auto task = pop_res.first;
                // hop num
                blockTask[0] = task.first;
                // row_num
                blockTask[1] = task.second;
            }
        }
        __syncthreads();
//        if (!sharedRes[0] && task_queue.empty())
        if (!sharedRes[0])
            break;
        // result.size() > num_rows * 5 just for test, should have a better check policy
//        if (!sharedRes[0] && result.size() > num_rows * 5)
//            break;
//        else if (!sharedRes[0])
//            continue;
        // run task, same block threads have same task(hop_num, row_num)
        const short hop_num = blockTask[0];
        const int64_t row = blockTask[1];
        const int64_t in_row_start = in_ptr[row];
        const int64_t deg = in_ptr[row + 1] - in_row_start;

        if (deg <= num_picks[hop_num - 1]) {
//            std::printf("row: %ld, deg: %ld, num_picks: %ld\n", row, deg, num_picks[hop_num - 1]);
            // just copy row when there is not enough nodes to sample
            for (int idx = threadIdx.x; idx < deg; idx += BLOCK_SIZE) {
                const int64_t in_idx = in_row_start + idx;
                result.push_back({hop_num, row, in_index[in_idx], data ? data[in_idx] : in_idx});
//                std::printf("result push hop_num: %d, row: %ld, col: %ld, data: %ld\n", hop_num, row, in_index[in_idx], data ? data[in_idx] : in_idx);
                // last hop don't need to push task
                if (hop_num < hops) {
                    if (!bits[in_index[in_idx] + total_num_rows * (hop_num - 1)]) {
//                        auto old = bits.set(in_index[in_idx] * hop_num);
                        auto old = atomicOr(&bits[in_index[in_idx] + total_num_rows * (hop_num - 1)], 1);
                        if (!old) {
                            task_queue.push({hop_num + 1, in_index[in_idx]});
                        }
                    }
                }
            }
        } else {
            // generate permutation list via reservoir algorithm
            // reservoir init
            for (int idx = threadIdx.x; idx < num_picks[hop_num - 1]; idx += BLOCK_SIZE) {
                permList[idx] = idx;
            }
            __syncthreads();

            for (int idx = num_picks[hop_num - 1] + threadIdx.x; idx < deg; idx += BLOCK_SIZE) {
                const int num = hiprand(&rng) % (idx + 1);
                if (num < num_picks[hop_num - 1]) {
                    // use shared memory, faster than DGL?
                    AtomicMax(permList + num, idx);
                }
            }
            __syncthreads();

            for (int idx = threadIdx.x; idx < num_picks[hop_num - 1]; idx += BLOCK_SIZE) {
                // permList[idx] is the idx of the sampled edge, from 0 to deg-1, should be added with in_row_start
                const int64_t perm_idx = permList[idx] + in_row_start;
                result.push_back({hop_num, row, in_index[perm_idx], data ? data[perm_idx] : perm_idx});
//                std::printf("result push hop_num: %d, row: %ld, col: %ld, data: %ld\n", hop_num, row, in_index[perm_idx], data ? data[perm_idx] : perm_idx);
                // last hop don't need to push task
                if (hop_num < hops) {
                    if (!bits[in_index[perm_idx] + total_num_rows * (hop_num - 1)]) {
                        auto old = atomicOr(&bits[in_index[perm_idx] + total_num_rows * (hop_num - 1)], 1);
                        if (!old) {
                            task_queue.push({hop_num + 1, in_index[perm_idx]});
                        }
                    }
                }
            }
        }
        // push self
        if (threadIdx.x == 0 && hop_num < hops) {
            if (!bits[row + total_num_rows * (hop_num - 1)]) {
                auto old = atomicOr(&bits[row + total_num_rows * (hop_num - 1)], 1);
                if (!old) {
                    task_queue.push({hop_num + 1, row});
                }
            }
        }
//        __syncthreads();
    }
}

/**
 * @brief Perform row-wise uniform sampling on a CSR matrix,
 * and generate a COO matrix, without replacement.
 *
 * @tparam IdType The ID type used for matrices.
 * @tparam TILE_SIZE The number of rows covered by each threadblock.
 * @param rand_seed The random seed to use.
 * @param num_picks The number of non-zeros to pick per row.
 * @param num_rows The number of rows to pick.
 * @param in_rows The set of rows to pick.
 * @param in_ptr The indptr array of the input CSR.
 * @param in_index The indices array of the input CSR.
 * @param data The data array of the input CSR.
 * @param out_ptr The offset to write each row to in the output COO.
 * @param out_rows The rows of the output COO (output).
 * @param out_cols The columns of the output COO (output).
 * @param out_idxs The data array of the output COO (output).
 */
template <typename IdType, int TILE_SIZE>
__global__ void _CSRRowWiseSampleUniformKernel(
    const uint64_t rand_seed, const int64_t num_picks, const int64_t num_rows,
    const IdType* const in_rows, const IdType* const in_ptr,
    const IdType* const in_index, const IdType* const data,
    const IdType* const out_ptr, IdType* const out_rows, IdType* const out_cols,
    IdType* const out_idxs) {
  // we assign one warp per row
  assert(blockDim.x == BLOCK_SIZE);

  int64_t out_row = blockIdx.x * TILE_SIZE;
  const int64_t last_row =
      min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_rows);

  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);
//    hiprand_init(rand_seed, 0, 0, &rng);

  while (out_row < last_row) {
    const int64_t row = in_rows[out_row];
    const int64_t in_row_start = in_ptr[row];
    const int64_t deg = in_ptr[row + 1] - in_row_start;
    const int64_t out_row_start = out_ptr[out_row];

    if (deg <= num_picks) {
      // just copy row when there is not enough nodes to sample.
      for (int idx = threadIdx.x; idx < deg; idx += BLOCK_SIZE) {
        const IdType in_idx = in_row_start + idx;
        out_rows[out_row_start + idx] = row;
        out_cols[out_row_start + idx] = in_index[in_idx];
        out_idxs[out_row_start + idx] = data ? data[in_idx] : in_idx;
      }
    } else {
      // generate permutation list via reservoir algorithm
      for (int idx = threadIdx.x; idx < num_picks; idx += BLOCK_SIZE) {
        out_idxs[out_row_start + idx] = idx;
      }
      __syncthreads();

      for (int idx = num_picks + threadIdx.x; idx < deg; idx += BLOCK_SIZE) {
        const int num = hiprand(&rng) % (idx + 1);
        if (num < num_picks) {
          // use max so as to achieve the replacement order the serial
          // algorithm would have
          AtomicMax(out_idxs + out_row_start + num, idx);
        }
      }
      __syncthreads();

      // copy permutation over
      for (int idx = threadIdx.x; idx < num_picks; idx += BLOCK_SIZE) {
        const IdType perm_idx = out_idxs[out_row_start + idx] + in_row_start;
        out_rows[out_row_start + idx] = row;
        out_cols[out_row_start + idx] = in_index[perm_idx];
        out_idxs[out_row_start + idx] = data ? data[perm_idx] : perm_idx;
      }
    }
    out_row += 1;
  }
}

/**
 * @brief Perform row-wise uniform sampling on a CSR matrix,
 * and generate a COO matrix, with replacement.
 *
 * @tparam IdType The ID type used for matrices.
 * @tparam TILE_SIZE The number of rows covered by each threadblock.
 * @param rand_seed The random seed to use.
 * @param num_picks The number of non-zeros to pick per row.
 * @param num_rows The number of rows to pick.
 * @param in_rows The set of rows to pick.
 * @param in_ptr The indptr array of the input CSR.
 * @param in_index The indices array of the input CSR.
 * @param data The data array of the input CSR.
 * @param out_ptr The offset to write each row to in the output COO.
 * @param out_rows The rows of the output COO (output).
 * @param out_cols The columns of the output COO (output).
 * @param out_idxs The data array of the output COO (output).
 */
template <typename IdType, int TILE_SIZE>
__global__ void _CSRRowWiseSampleUniformReplaceKernel(
    const uint64_t rand_seed, const int64_t num_picks, const int64_t num_rows,
    const IdType* const in_rows, const IdType* const in_ptr,
    const IdType* const in_index, const IdType* const data,
    const IdType* const out_ptr, IdType* const out_rows, IdType* const out_cols,
    IdType* const out_idxs) {
  // we assign one warp per row
  assert(blockDim.x == BLOCK_SIZE);

  int64_t out_row = blockIdx.x * TILE_SIZE;
  const int64_t last_row =
      min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_rows);

  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);

  while (out_row < last_row) {
    const int64_t row = in_rows[out_row];
    const int64_t in_row_start = in_ptr[row];
    const int64_t out_row_start = out_ptr[out_row];
    const int64_t deg = in_ptr[row + 1] - in_row_start;

    if (deg > 0) {
      // each thread then blindly copies in rows only if deg > 0.
      for (int idx = threadIdx.x; idx < num_picks; idx += BLOCK_SIZE) {
        const int64_t edge = hiprand(&rng) % deg;
        const int64_t out_idx = out_row_start + idx;
        out_rows[out_idx] = row;
        out_cols[out_idx] = in_index[in_row_start + edge];
        out_idxs[out_idx] =
            data ? data[in_row_start + edge] : in_row_start + edge;
      }
    }
    out_row += 1;
  }
}

}  // namespace

///////////////////////////// CSR sampling //////////////////////////

template <DGLDeviceType XPU, typename IdType>
COOMatrix _CSRRowWiseSamplingUniform(
    CSRMatrix mat, IdArray rows, const int64_t num_picks, const bool replace) {
  const auto& ctx = rows->ctx;
  auto device = runtime::DeviceAPI::Get(ctx);
  hipStream_t stream = runtime::getCurrentCUDAStream();

  const int64_t num_rows = rows->shape[0];
  const IdType* const slice_rows = static_cast<const IdType*>(rows->data);

  IdArray picked_row =
      NewIdArray(num_rows * num_picks, ctx, sizeof(IdType) * 8);
  IdArray picked_col =
      NewIdArray(num_rows * num_picks, ctx, sizeof(IdType) * 8);
  IdArray picked_idx =
      NewIdArray(num_rows * num_picks, ctx, sizeof(IdType) * 8);
  IdType* const out_rows = static_cast<IdType*>(picked_row->data);
  IdType* const out_cols = static_cast<IdType*>(picked_col->data);
  IdType* const out_idxs = static_cast<IdType*>(picked_idx->data);

  const IdType* in_ptr = static_cast<IdType*>(GetDevicePointer(mat.indptr));
  const IdType* in_cols = static_cast<IdType*>(GetDevicePointer(mat.indices));
  const IdType* data = CSRHasData(mat)
                           ? static_cast<IdType*>(GetDevicePointer(mat.data))
                           : nullptr;

  // compute degree
  IdType* out_deg = static_cast<IdType*>(
      device->AllocWorkspace(ctx, (num_rows + 1) * sizeof(IdType)));
  if (replace) {
    const dim3 block(512);
    const dim3 grid((num_rows + block.x - 1) / block.x);
    CUDA_KERNEL_CALL(
        _CSRRowWiseSampleDegreeReplaceKernel, grid, block, 0, stream, num_picks,
        num_rows, slice_rows, in_ptr, out_deg);
  } else {
    const dim3 block(512);
    const dim3 grid((num_rows + block.x - 1) / block.x);
    CUDA_KERNEL_CALL(
        _CSRRowWiseSampleDegreeKernel, grid, block, 0, stream, num_picks,
        num_rows, slice_rows, in_ptr, out_deg);
  }

  // fill out_ptr
  IdType* out_ptr = static_cast<IdType*>(
      device->AllocWorkspace(ctx, (num_rows + 1) * sizeof(IdType)));
  size_t prefix_temp_size = 0;
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      nullptr, prefix_temp_size, out_deg, out_ptr, num_rows + 1, stream));
  void* prefix_temp = device->AllocWorkspace(ctx, prefix_temp_size);
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      prefix_temp, prefix_temp_size, out_deg, out_ptr, num_rows + 1, stream));
  device->FreeWorkspace(ctx, prefix_temp);
  device->FreeWorkspace(ctx, out_deg);

  hipEvent_t copyEvent;
  CUDA_CALL(hipEventCreate(&copyEvent));

  // TODO(dlasalle): use pinned memory to overlap with the actual sampling, and
  // wait on a cudaevent
  IdType new_len;
  // copy using the internal current stream
  device->CopyDataFromTo(
      out_ptr, num_rows * sizeof(new_len), &new_len, 0, sizeof(new_len), ctx,
      DGLContext{kDGLCPU, 0}, mat.indptr->dtype);
  CUDA_CALL(hipEventRecord(copyEvent, stream));

//  const uint64_t random_seed = RandomEngine::ThreadLocal()->RandInt(1000000000);
  // fix only for reproduce!
  const uint64_t random_seed = 1234;

  // select edges
  // the number of rows each thread block will cover
  constexpr int TILE_SIZE = 128 / BLOCK_SIZE;
  if (replace) {  // with replacement
    const dim3 block(BLOCK_SIZE);
    const dim3 grid((num_rows + TILE_SIZE - 1) / TILE_SIZE);
    CUDA_KERNEL_CALL(
        (_CSRRowWiseSampleUniformReplaceKernel<IdType, TILE_SIZE>), grid, block,
        0, stream, random_seed, num_picks, num_rows, slice_rows, in_ptr,
        in_cols, data, out_ptr, out_rows, out_cols, out_idxs);
  } else {  // without replacement
    const dim3 block(BLOCK_SIZE);
    const dim3 grid((num_rows + TILE_SIZE - 1) / TILE_SIZE);
    CUDA_KERNEL_CALL(
        (_CSRRowWiseSampleUniformKernel<IdType, TILE_SIZE>), grid, block, 0,
        stream, random_seed, num_picks, num_rows, slice_rows, in_ptr, in_cols,
        data, out_ptr, out_rows, out_cols, out_idxs);
  }
  device->FreeWorkspace(ctx, out_ptr);

  // wait for copying `new_len` to finish
  CUDA_CALL(hipEventSynchronize(copyEvent));
  CUDA_CALL(hipEventDestroy(copyEvent));

  // from shape{num_rows * num_picks} to shape{new_len}
  // I guess: if wait the copy of `new_len`, then new IdArray picked_row, picked_col, picked_idx with shape `new_len`, not need to reshape here
  // DGL use async copy, overlap copy with the sampling kernel execution
  // and the kernel need the data of picked_row, picked_col, picked_idx, so need to new them with enough shape before and reshape here
  picked_row = picked_row.CreateView({new_len}, picked_row->dtype);
  picked_col = picked_col.CreateView({new_len}, picked_col->dtype);
  picked_idx = picked_idx.CreateView({new_len}, picked_idx->dtype);

  return COOMatrix(
      mat.num_rows, mat.num_cols, picked_row, picked_col, picked_idx);
}

template <DGLDeviceType XPU, typename IdType>
COOMatrix CSRRowWiseSamplingUniform(
    CSRMatrix mat, IdArray rows, const int64_t num_picks, const bool replace) {
  if (num_picks == -1) {
    // Basically this is UnitGraph::InEdges().
    COOMatrix coo = CSRToCOO(CSRSliceRows(mat, rows), false);
    IdArray sliced_rows = IndexSelect(rows, coo.row);
    return COOMatrix(
        mat.num_rows, mat.num_cols, sliced_rows, coo.col, coo.data);
  } else {
    return _CSRRowWiseSamplingUniform<XPU, IdType>(
        mat, rows, num_picks, replace);
  }
}

stdgpu::vector<selectedEdgeInfo> res_vector;
//stdgpu::queue<thrust::pair<short, int64_t>> task_queue;
const int RES_VEC_CAP = 1e8;
stdgpu::index_t vector_cap = 0;
bool first_time = true;
int64_t old_size = 0;

std::vector<COOMatrix> CustomCSRRowWiseSamplingUniformTaskParallelism(
        CSRMatrix mat, IdArray rows, const IdArray &num_picks) {
//    std::printf("CustomCSRRowWiseSamplingUniformTaskParallelism run here\n");
    const auto& ctx = rows->ctx;
    const auto& num_picks_vec = num_picks.ToVector<int64_t>();
    auto device = runtime::DeviceAPI::Get(ctx);
    hipStream_t stream = runtime::getCurrentCUDAStream();

    // 1-hop seed nodes number
    const int64_t num_rows = rows->shape[0];
    const auto hops = num_picks->shape[0];

    // rows(i.e. batch nodes id) static cast to int64_t array, also a device ptr
    const int64_t* const sliced_rows = static_cast<const int64_t*>(rows->data);

    // for COO result
    // picked_row, picked_col, picked_idx: IdArray(NdArray)
    // out_row, out_col, out_idx: The data pointer points to the allocated data space(here is device ptr)
    std::vector<IdArray> picked_rows(hops), picked_cols(hops), picked_indices(hops);
    const int64_t* in_ptr = static_cast<int64_t*>(GetDevicePointer(mat.indptr));
    const int64_t* in_cols = static_cast<int64_t*>(GetDevicePointer(mat.indices));
    const int64_t* data = CSRHasData(mat)
                          ? static_cast<int64_t*>(GetDevicePointer(mat.data))
                          : nullptr;
    const int64_t* num_picks_ptr = static_cast<int64_t*>(GetDevicePointer(num_picks));

    // allocate space for stdgpu container
    stdgpu::index_t queue_cap = num_rows;
    // last hop sample result do not need to enqueue
    for (int i = 0; i < hops - 1; i++)
        queue_cap += queue_cap * (num_picks_vec[i] + 1);
    // pair(hop_num, src_node_id)
    nvtxRangePushA("create task_queue");
    auto task_queue = stdgpu::queue<thrust::pair<short, int64_t>>::createDeviceObject(queue_cap);
    nvtxRangePop();

    nvtxRangePushA("create bits");
    uint* bool_arr = static_cast<uint *>(device->AllocWorkspace(ctx, mat.num_rows * (hops - 1) * sizeof(uint)));
    CUDA_CALL(hipMemset(bool_arr, 0, mat.num_rows * (hops - 1) * sizeof(uint)));
    nvtxRangePop();

    // init
    const dim3 init_block(512);
    const dim3 init_grid((num_rows + init_block.x - 1) / init_block.x);
//    const dim3 init_grid((num_rows * hops + init_block.x - 1) / init_block.x);
    CUDA_KERNEL_CALL((queue_init), init_grid, init_block, 0, stream, task_queue, bool_arr, sliced_rows, num_rows, hops, mat.num_rows);
//    assert(task_queue.size() == num_rows);

    nvtxRangePushA("create or clear container");
    if (first_time) {
        int64_t node_num = num_rows;
//        vector_cap = num_rows * num_picks_vec[0];
        // last hop sample result need to push into result vector
        for (int i = 0; i < hops; i++) {
            // the dstnodes of current hop should be the srcnodes of next hop
            vector_cap += node_num * num_picks_vec[i];
            node_num += vector_cap;
        }
        res_vector = stdgpu::vector<selectedEdgeInfo>::createDeviceObject(RES_VEC_CAP);

//        nvtxRangePushA("create bits");
//        bits = stdgpu::bitset<>::createDeviceObject(static_cast<stdgpu::index_t>(mat.num_rows * (hops - 1)));
//        nvtxRangePop();

        first_time = false;
    }
        // task queue will be "clear" when sample kernel finished
    else{
        if (old_size + vector_cap >= RES_VEC_CAP){
            res_vector.clear();
            old_size = 0;
        }
    }
    nvtxRangePop();

//    const uint64_t random_seed = RandomEngine::ThreadLocal()->RandInt(1000000000);
    // fix only for reproduce!
    const uint64_t random_seed = 1234;

    const dim3 block(BLOCK_SIZE);
    // should gird num be max?
    // best performance:arxiv, [25,10]
    const dim3 grid(num_rows);
//    const dim3 grid(num_rows * hops);
//    const dim3 grid(1);

//    std::printf("queue valid:%d\n", task_queue.valid());
    CUDA_KERNEL_CALL((_CSRRowWiseSampleUniformTaskParallelismKernel), grid, block, 0, stream,
                     random_seed, num_picks_ptr, sliced_rows, num_rows, hops, mat.num_rows, in_ptr, in_cols, data,
                     task_queue, bool_arr, res_vector);
//    std::printf("queue valid:%d\n", task_queue.valid());
//    assert(task_queue.empty());
//    std::printf("cuda kernel finished\n");

    // 传多个COO res的row, col, idx的指针的指针，用res_vector取fill，逻辑上最直观. 传指针的指针要写个demo试一下
    nvtxRangePushA("get result");
    int64_t node_num = num_rows;
    nvtxRangePushA("NewIdArray");
    for(int i = 0; i < hops; i++) {
        int64_t hop_res_num = node_num * num_picks_vec[i];
        picked_rows[i] = NewIdArray(hop_res_num, ctx, sizeof(int64_t) * 8);
        picked_cols[i] = NewIdArray(hop_res_num, ctx, sizeof(int64_t) * 8);
        picked_indices[i] = NewIdArray(hop_res_num, ctx, sizeof(int64_t) * 8);
        node_num += hop_res_num;
    }
    nvtxRangePop();

    std::vector<COOMatrix> ret_coo(hops);

    auto range_vec = res_vector.device_range();
    auto zip_begin = thrust::make_zip_iterator(thrust::make_tuple(
            thrust::make_transform_iterator(range_vec.begin(), RowTrans()),
            thrust::make_transform_iterator(range_vec.begin(), ColTrans()),
            thrust::make_transform_iterator(range_vec.begin(), DataTrans())
    ));
    auto zip_end = thrust::make_zip_iterator(thrust::make_tuple(
            thrust::make_transform_iterator(range_vec.end(), RowTrans()),
            thrust::make_transform_iterator(range_vec.end(), ColTrans()),
            thrust::make_transform_iterator(range_vec.end(), DataTrans())
    ));
//    hipPointerAttribute_t attributes;
    for(int i = 0; i < hops; i++) {
        auto out_rows = static_cast<int64_t*>(picked_rows[i]->data);
        auto out_cols = static_cast<int64_t*>(picked_cols[i]->data);
        auto out_idx = static_cast<int64_t*>(picked_indices[i]->data);
        nvtxRangePushA("thrust process");
//        CUDA_CALL(hipPointerGetAttributes(&attributes, out_rows));
//        if (attributes.type == hipMemoryTypeDevice)
//        {
//            std::cout << "Device pointer is a valid device pointer." << std::endl;
//        }

        auto zip_res = thrust::make_zip_iterator(thrust::make_tuple(
                out_rows,
                out_cols,
                out_idx
        ));
        int64_t new_size = thrust::copy_if(
                thrust::device,
                zip_begin + old_size,
                zip_end,
                thrust::make_transform_iterator(range_vec.begin() + old_size, HopTrans()),
                zip_res,
                CheckHopNum(i + 1)
                ) - zip_res;
        nvtxRangePop();

        nvtxRangePushA("CreateView");
        picked_rows[i] = picked_rows[i].CreateView({new_size}, picked_rows[i]->dtype);
        picked_cols[i] = picked_cols[i].CreateView({new_size}, picked_cols[i]->dtype);
        picked_indices[i] = picked_indices[i].CreateView({new_size}, picked_indices[i]->dtype);
        nvtxRangePop();
        ret_coo[i] = COOMatrix(mat.num_rows, mat.num_cols, picked_rows[i], picked_cols[i], picked_indices[i]);
    }
    nvtxRangePop();

    old_size = res_vector.size();

//    nvtxRangePushA("free res_vector");
    //TODO: should free them once when program ends(last iteration)... but I have not found an easy way to do that.
//    stdgpu::vector<selectedEdgeInfo>::destroyDeviceObject(res_vector);
//    nvtxRangePop();
//    std::printf("CustomCSRRowWiseSamplingUniformTaskParallelism finished here\n");
    device->FreeWorkspace(ctx, bool_arr);
    stdgpu::queue<thrust::pair<short, int64_t>>::destroyDeviceObject(task_queue);
//    stdgpu::bitset<>::destroyDeviceObject(bits);
    return ret_coo;
}

template COOMatrix CSRRowWiseSamplingUniform<kDGLCUDA, int32_t>(
    CSRMatrix, IdArray, int64_t, bool);
template COOMatrix CSRRowWiseSamplingUniform<kDGLCUDA, int64_t>(
    CSRMatrix, IdArray, int64_t, bool);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
