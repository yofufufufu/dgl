#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2021 by Contributors
 * @file array/cuda/rowwise_sampling.cu
 * @brief uniform rowwise sampling
 */

#include <hiprand/hiprand_kernel.h>
#include <dgl/random.h>
#include <dgl/runtime/device_api.h>

#include <nvtx3/nvToolsExt.h>

#include "../../array/cuda/atomic.cuh"
#include "../../runtime/cuda/cuda_common.h"
#include "./dgl_cub.cuh"
#include "./utils.h"

namespace dgl {
using namespace cuda;
using namespace aten::cuda;
namespace aten {
namespace impl {

namespace {

constexpr int BLOCK_SIZE = 128;
constexpr int BLOCK_SIZE_CUSTOM = 96;

/**
 * @brief Compute the size of each row in the sampled CSR, without replacement.
 *
 * @tparam IdType The type of node and edge indexes.
 * @param num_picks The number of non-zero entries to pick per row.
 * @param num_rows The number of rows to pick.
 * @param in_rows The set of rows to pick.
 * @param in_ptr The index where each row's edges start.
 * @param out_deg The size of each row in the sampled matrix, as indexed by
 * `in_rows` (output).
 */
template <typename IdType>
__global__ void _CSRRowWiseSampleDegreeKernel(
    const int64_t num_picks, const int64_t num_rows,
    const IdType* const in_rows, const IdType* const in_ptr,
    IdType* const out_deg) {
  const int tIdx = threadIdx.x + blockIdx.x * blockDim.x;

  if (tIdx < num_rows) {
    const int in_row = in_rows[tIdx];
    const int out_row = tIdx;
    out_deg[out_row] = min(
        static_cast<IdType>(num_picks), in_ptr[in_row + 1] - in_ptr[in_row]);

    if (out_row == num_rows - 1) {
      // make the prefixsum work
      out_deg[num_rows] = 0;
    }
  }
}

/**
 * @brief Compute the size of each row in the sampled CSR, with replacement.
 *
 * @tparam IdType The type of node and edge indexes.
 * @param num_picks The number of non-zero entries to pick per row.
 * @param num_rows The number of rows to pick.
 * @param in_rows The set of rows to pick.
 * @param in_ptr The index where each row's edges start.
 * @param out_deg The size of each row in the sampled matrix, as indexed by
 * `in_rows` (output).
 */
template <typename IdType>
__global__ void _CSRRowWiseSampleDegreeReplaceKernel(
    const int64_t num_picks, const int64_t num_rows,
    const IdType* const in_rows, const IdType* const in_ptr,
    IdType* const out_deg) {
  const int tIdx = threadIdx.x + blockIdx.x * blockDim.x;

  if (tIdx < num_rows) {
    const int64_t in_row = in_rows[tIdx];
    const int64_t out_row = tIdx;

    if (in_ptr[in_row + 1] - in_ptr[in_row] == 0) {
      out_deg[out_row] = 0;
    } else {
      out_deg[out_row] = static_cast<IdType>(num_picks);
    }

    if (out_row == num_rows - 1) {
      // make the prefixsum work
      out_deg[num_rows] = 0;
    }
  }
}

// the sample result of a hop
//struct selectedEdgeInfo {
//    short hop;
//    int64_t row_num;
//    int64_t col_num;
//    int64_t data_num;
//};
struct selectedEdgeInfo {
    int64_t* rows;
    int64_t* cols;
    int64_t* datas;
};

// must be volatile!
// __device__ volatile uint tail_index;
 __device__ uint tail_index;
// __device__ volatile uint finished_block_num;
 __device__ uint task_idx;
// should try to push the node with more edges first
//TODO: if let init_kernel kernel do more things, try overlap it with more host codes
__global__ void init_kernel(
        thrust::pair<short, int64_t>* queue, uint* bits, const int64_t* const in_rows,
        const int64_t num_rows, const int hops, const int64_t queue_cap) {
    const int tIdx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tIdx == 0) {
        tail_index = num_rows;
//        finished_block_num = 0;
        task_idx = 0;
    }
//    if (tIdx < num_rows * hops) {
//        int hop_num = 1 + tIdx / num_rows;
//        queue.push({hop_num, in_rows[tIdx % num_rows]});
//        if (hop_num > 1)
//            bits[in_rows[tIdx % num_rows] + total_num_rows * (hop_num - 2)] = 1;
//    }
    if (tIdx < num_rows) {
        queue[tIdx] = {1, in_rows[tIdx]};
    }
    // thread num > queue_cap, should restrict else condition
    else if (tIdx < queue_cap) {
        // avoid read uninitialized value, which can cause strange error
        queue[tIdx] = {0, -1};
    }
}

__launch_bounds__(BLOCK_SIZE_CUSTOM) __global__ void _CSRRowWiseSampleUniformTaskParallelismKernel(
        const uint64_t rand_seed, const int64_t * num_picks, uint* vector_lens,
        const int64_t num_rows, const int hops, const int64_t total_num_rows,
        const int64_t * const in_ptr, const int64_t * const in_index, const int64_t * const data,
        // must be volatile!
        volatile thrust::pair<short, int64_t>* task_queue,
        uint* bits,
        selectedEdgeInfo* result
        ) {
//    __shared__ int64_t blockTask[2];
    __shared__ bool sharedRes[1];
    __shared__ uint sharedTask[1];
    // num_pick cannot be larger than 128
    // any better solution?
    __shared__ int64_t permList[128];

    // writes to tail_index and finished_block_num is not visible(at least within a period of time) to other blocks
    // if variables are not volatile, the value will be cached?
    // so other blocks will loop infinite time in a very short time, which causes program hung.
    // `printf` can change data visibility by some ways...
    // see: https://forums.developer.nvidia.com/t/printf-in-cuda-kernel-changes-program-behavior/234524

//    if (threadIdx.x == 0) {
//        sharedRes[0] = false;
//        while (finished_block_num != tail_index) {
//            if (blockIdx.x < tail_index) {
//                sharedRes[0] = true;
//                break;
//            }
//        }
//    }
//    __syncthreads();

    if (threadIdx.x == 0) {
        auto taskid = atomicAdd(&task_idx, 1);
        sharedTask[0] = taskid;
        if (taskid >= tail_index) {
            // theoretical max block resident on 3090 when block size 96, and training need drop last when batchSize < 1312
            if (taskid < 1312)
                sharedRes[0] = true;
            else
                sharedRes[0] = false;
        }
        else
            sharedRes[0] = true;
    }
    __syncthreads();

    if (sharedRes[0]) {
        hiprandStatePhilox4_32_10_t rng;
        // different block has different seed
        // different thread in block has different (sub)sequence
        hiprand_init(rand_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);
        // hiprand_init(rand_seed, 0, 0, &rng);

        // run task, same block threads have same task(hop_num, row_num)
        // write to task queue may not visible, so task will be init_kernel value, i.e.{0,-1}
        // just do again, use while loop(any better solution?).
        // `task_queue` also must be volatile, or it will be cached, causing loop infinite time
        auto const taskid = sharedTask[0];
        short hop_num = task_queue[taskid].first;
        int64_t row = task_queue[taskid].second;
        while (hop_num == 0 || row == -1) {
            hop_num = task_queue[taskid].first;
            row = task_queue[taskid].second;
        }

        const int64_t in_row_start = in_ptr[row];
        const int64_t deg = in_ptr[row + 1] - in_row_start;
        const int64_t num_pick = num_picks[hop_num - 1];

        if (deg <= num_pick) {
            // just copy row when there is not enough nodes to sample
            for (int idx = threadIdx.x; idx < deg; idx += BLOCK_SIZE_CUSTOM) {
                const int64_t in_idx = in_row_start + idx;
                const int64_t neighbor = in_index[in_idx];
                auto index = atomicAdd(vector_lens + (hop_num - 1), 1);
                result[hop_num - 1].rows[index] = row;
                result[hop_num - 1].cols[index] = neighbor;
                result[hop_num - 1].datas[index] = data ? data[in_idx] : in_idx;
                // last hop don't need to push task
                // TODO: can this atomic operation be optimized?
                if (hop_num < hops) {
                    const int64_t bits_offset = neighbor + total_num_rows * (hop_num - 1);
                    if (!bits[bits_offset]) {
                        //  auto old = bits.set(in_index[in_idx] * hop_num);
                        auto old = atomicOr(bits + bits_offset, 1);
                        if (!old) {
                            auto tail = atomicAdd(&tail_index, 1);
                            task_queue[tail].first = hop_num + 1;
                            task_queue[tail].second = neighbor;
//                                task_queue[tail] = {hop_num + 1, in_index[in_idx]};
                        }
                    }
                }
            }
        } else {
            // generate permutation list via reservoir algorithm
            // reservoir init
            for (int idx = threadIdx.x; idx < num_pick; idx += BLOCK_SIZE_CUSTOM) {
                permList[idx] = idx;
            }
            __syncthreads();

            for (int idx = num_pick + threadIdx.x; idx < deg; idx += BLOCK_SIZE_CUSTOM) {
                const int num = hiprand(&rng) % (idx + 1);
                if (num < num_pick) {
                    // use shared memory, faster than DGL?
                    AtomicMax(permList + num, idx);
                }
            }
            __syncthreads();

            for (int idx = threadIdx.x; idx < num_pick; idx += BLOCK_SIZE_CUSTOM) {
                // permList[idx] is the idx of the sampled edge, from 0 to deg-1, should be added with in_row_start
                const int64_t perm_idx = permList[idx] + in_row_start;
                const int64_t neighbor = in_index[perm_idx];
                auto index = atomicAdd(vector_lens + (hop_num - 1), 1);
                result[hop_num - 1].rows[index] = row;
                result[hop_num - 1].cols[index] = neighbor;
                result[hop_num - 1].datas[index] = data ? data[perm_idx] : perm_idx;
                // last hop don't need to push task
                if (hop_num < hops) {
                    const int64_t bits_offset = neighbor + total_num_rows * (hop_num - 1);
                    if (!bits[bits_offset]) {
                        auto old = atomicOr(bits + bits_offset, 1);
                        if (!old) {
                            auto tail = atomicAdd(&tail_index, 1);
//                                task_queue[tail] = {hop_num + 1, in_index[perm_idx]};
                            task_queue[tail].first = hop_num + 1;
                            task_queue[tail].second = neighbor;
                        }
                    }
                }
            }
        }
        // push self
        if (threadIdx.x == 0 && hop_num < hops) {
            const int64_t bits_offset = row + total_num_rows * (hop_num - 1);
            if (!bits[bits_offset]) {
                auto old = atomicOr(bits + bits_offset, 1);
                if (!old) {
                    auto tail = atomicAdd(&tail_index, 1);
//                        task_queue[tail] = {hop_num + 1, row};
                    task_queue[tail].first = hop_num + 1;
                    task_queue[tail].second = row;
                }
            }
        }
        // enough blocks, every block loop once.
    } else {
        return;
    }
}

/**
 * @brief Perform row-wise uniform sampling on a CSR matrix,
 * and generate a COO matrix, without replacement.
 *
 * @tparam IdType The ID type used for matrices.
 * @tparam TILE_SIZE The number of rows covered by each threadblock.
 * @param rand_seed The random seed to use.
 * @param num_picks The number of non-zeros to pick per row.
 * @param num_rows The number of rows to pick.
 * @param in_rows The set of rows to pick.
 * @param in_ptr The indptr array of the input CSR.
 * @param in_index The indices array of the input CSR.
 * @param data The data array of the input CSR.
 * @param out_ptr The offset to write each row to in the output COO.
 * @param out_rows The rows of the output COO (output).
 * @param out_cols The columns of the output COO (output).
 * @param out_idxs The data array of the output COO (output).
 */
template <typename IdType, int TILE_SIZE>
__global__ void _CSRRowWiseSampleUniformKernel(
    const uint64_t rand_seed, const int64_t num_picks, const int64_t num_rows,
    const IdType* const in_rows, const IdType* const in_ptr,
    const IdType* const in_index, const IdType* const data,
    const IdType* const out_ptr, IdType* const out_rows, IdType* const out_cols,
    IdType* const out_idxs) {
  // we assign one warp per row
  assert(blockDim.x == BLOCK_SIZE);

  int64_t out_row = blockIdx.x * TILE_SIZE;
  const int64_t last_row =
      min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_rows);

  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);
//    hiprand_init(rand_seed, 0, 0, &rng);

  while (out_row < last_row) {
    const int64_t row = in_rows[out_row];
    const int64_t in_row_start = in_ptr[row];
    const int64_t deg = in_ptr[row + 1] - in_row_start;
    const int64_t out_row_start = out_ptr[out_row];

    if (deg <= num_picks) {
      // just copy row when there is not enough nodes to sample.
      for (int idx = threadIdx.x; idx < deg; idx += BLOCK_SIZE) {
        const IdType in_idx = in_row_start + idx;
        out_rows[out_row_start + idx] = row;
        out_cols[out_row_start + idx] = in_index[in_idx];
        out_idxs[out_row_start + idx] = data ? data[in_idx] : in_idx;
      }
    } else {
      // generate permutation list via reservoir algorithm
      for (int idx = threadIdx.x; idx < num_picks; idx += BLOCK_SIZE) {
        out_idxs[out_row_start + idx] = idx;
      }
      __syncthreads();

      for (int idx = num_picks + threadIdx.x; idx < deg; idx += BLOCK_SIZE) {
        const int num = hiprand(&rng) % (idx + 1);
        if (num < num_picks) {
          // use max so as to achieve the replacement order the serial
          // algorithm would have
          AtomicMax(out_idxs + out_row_start + num, idx);
        }
      }
      __syncthreads();

      // copy permutation over
      for (int idx = threadIdx.x; idx < num_picks; idx += BLOCK_SIZE) {
        const IdType perm_idx = out_idxs[out_row_start + idx] + in_row_start;
        out_rows[out_row_start + idx] = row;
        out_cols[out_row_start + idx] = in_index[perm_idx];
        out_idxs[out_row_start + idx] = data ? data[perm_idx] : perm_idx;
      }
    }
    out_row += 1;
  }
}

/**
 * @brief Perform row-wise uniform sampling on a CSR matrix,
 * and generate a COO matrix, with replacement.
 *
 * @tparam IdType The ID type used for matrices.
 * @tparam TILE_SIZE The number of rows covered by each threadblock.
 * @param rand_seed The random seed to use.
 * @param num_picks The number of non-zeros to pick per row.
 * @param num_rows The number of rows to pick.
 * @param in_rows The set of rows to pick.
 * @param in_ptr The indptr array of the input CSR.
 * @param in_index The indices array of the input CSR.
 * @param data The data array of the input CSR.
 * @param out_ptr The offset to write each row to in the output COO.
 * @param out_rows The rows of the output COO (output).
 * @param out_cols The columns of the output COO (output).
 * @param out_idxs The data array of the output COO (output).
 */
template <typename IdType, int TILE_SIZE>
__global__ void _CSRRowWiseSampleUniformReplaceKernel(
    const uint64_t rand_seed, const int64_t num_picks, const int64_t num_rows,
    const IdType* const in_rows, const IdType* const in_ptr,
    const IdType* const in_index, const IdType* const data,
    const IdType* const out_ptr, IdType* const out_rows, IdType* const out_cols,
    IdType* const out_idxs) {
  // we assign one warp per row
  assert(blockDim.x == BLOCK_SIZE);

  int64_t out_row = blockIdx.x * TILE_SIZE;
  const int64_t last_row =
      min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_rows);

  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);

  while (out_row < last_row) {
    const int64_t row = in_rows[out_row];
    const int64_t in_row_start = in_ptr[row];
    const int64_t out_row_start = out_ptr[out_row];
    const int64_t deg = in_ptr[row + 1] - in_row_start;

    if (deg > 0) {
      // each thread then blindly copies in rows only if deg > 0.
      for (int idx = threadIdx.x; idx < num_picks; idx += BLOCK_SIZE) {
        const int64_t edge = hiprand(&rng) % deg;
        const int64_t out_idx = out_row_start + idx;
        out_rows[out_idx] = row;
        out_cols[out_idx] = in_index[in_row_start + edge];
        out_idxs[out_idx] =
            data ? data[in_row_start + edge] : in_row_start + edge;
      }
    }
    out_row += 1;
  }
}

}  // namespace

///////////////////////////// CSR sampling //////////////////////////

template <DGLDeviceType XPU, typename IdType>
COOMatrix _CSRRowWiseSamplingUniform(
    CSRMatrix mat, IdArray rows, const int64_t num_picks, const bool replace) {
  const auto& ctx = rows->ctx;
  auto device = runtime::DeviceAPI::Get(ctx);
  hipStream_t stream = runtime::getCurrentCUDAStream();

  const int64_t num_rows = rows->shape[0];
  const IdType* const slice_rows = static_cast<const IdType*>(rows->data);

  IdArray picked_row =
      NewIdArray(num_rows * num_picks, ctx, sizeof(IdType) * 8);
  IdArray picked_col =
      NewIdArray(num_rows * num_picks, ctx, sizeof(IdType) * 8);
  IdArray picked_idx =
      NewIdArray(num_rows * num_picks, ctx, sizeof(IdType) * 8);
  IdType* const out_rows = static_cast<IdType*>(picked_row->data);
  IdType* const out_cols = static_cast<IdType*>(picked_col->data);
  IdType* const out_idxs = static_cast<IdType*>(picked_idx->data);

  const IdType* in_ptr = static_cast<IdType*>(GetDevicePointer(mat.indptr));
  const IdType* in_cols = static_cast<IdType*>(GetDevicePointer(mat.indices));
  const IdType* data = CSRHasData(mat)
                           ? static_cast<IdType*>(GetDevicePointer(mat.data))
                           : nullptr;

  // compute degree
  IdType* out_deg = static_cast<IdType*>(
      device->AllocWorkspace(ctx, (num_rows + 1) * sizeof(IdType)));
  if (replace) {
    const dim3 block(512);
    const dim3 grid((num_rows + block.x - 1) / block.x);
    CUDA_KERNEL_CALL(
        _CSRRowWiseSampleDegreeReplaceKernel, grid, block, 0, stream, num_picks,
        num_rows, slice_rows, in_ptr, out_deg);
  } else {
    const dim3 block(512);
    const dim3 grid((num_rows + block.x - 1) / block.x);
    CUDA_KERNEL_CALL(
        _CSRRowWiseSampleDegreeKernel, grid, block, 0, stream, num_picks,
        num_rows, slice_rows, in_ptr, out_deg);
  }

  // fill out_ptr
  IdType* out_ptr = static_cast<IdType*>(
      device->AllocWorkspace(ctx, (num_rows + 1) * sizeof(IdType)));
  size_t prefix_temp_size = 0;
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      nullptr, prefix_temp_size, out_deg, out_ptr, num_rows + 1, stream));
  void* prefix_temp = device->AllocWorkspace(ctx, prefix_temp_size);
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      prefix_temp, prefix_temp_size, out_deg, out_ptr, num_rows + 1, stream));
  device->FreeWorkspace(ctx, prefix_temp);
  device->FreeWorkspace(ctx, out_deg);

  hipEvent_t copyEvent;
  CUDA_CALL(hipEventCreate(&copyEvent));

  // TODO(dlasalle): use pinned memory to overlap with the actual sampling, and
  // wait on a cudaevent
  IdType new_len;
  // copy using the internal current stream
  device->CopyDataFromTo(
      out_ptr, num_rows * sizeof(new_len), &new_len, 0, sizeof(new_len), ctx,
      DGLContext{kDGLCPU, 0}, mat.indptr->dtype);
  CUDA_CALL(hipEventRecord(copyEvent, stream));

//  const uint64_t random_seed = RandomEngine::ThreadLocal()->RandInt(1000000000);
  // fix only for reproduce!
  const uint64_t random_seed = 1234;

  // select edges
  // the number of rows each thread block will cover
  constexpr int TILE_SIZE = 128 / BLOCK_SIZE;
  if (replace) {  // with replacement
    const dim3 block(BLOCK_SIZE);
    const dim3 grid((num_rows + TILE_SIZE - 1) / TILE_SIZE);
    CUDA_KERNEL_CALL(
        (_CSRRowWiseSampleUniformReplaceKernel<IdType, TILE_SIZE>), grid, block,
        0, stream, random_seed, num_picks, num_rows, slice_rows, in_ptr,
        in_cols, data, out_ptr, out_rows, out_cols, out_idxs);
  } else {  // without replacement
    const dim3 block(BLOCK_SIZE);
    const dim3 grid((num_rows + TILE_SIZE - 1) / TILE_SIZE);
    CUDA_KERNEL_CALL(
        (_CSRRowWiseSampleUniformKernel<IdType, TILE_SIZE>), grid, block, 0,
        stream, random_seed, num_picks, num_rows, slice_rows, in_ptr, in_cols,
        data, out_ptr, out_rows, out_cols, out_idxs);
  }
  device->FreeWorkspace(ctx, out_ptr);

  // wait for copying `new_len` to finish
  CUDA_CALL(hipEventSynchronize(copyEvent));
  CUDA_CALL(hipEventDestroy(copyEvent));

  // from shape{num_rows * num_picks} to shape{new_len}
  // I guess: if wait the copy of `new_len`, then new IdArray picked_row, picked_col, picked_idx with shape `new_len`, not need to reshape here
  // DGL use async copy, overlap copy with the sampling kernel execution
  // and the kernel need the data of picked_row, picked_col, picked_idx, so need to new them with enough shape before and reshape here
  picked_row = picked_row.CreateView({new_len}, picked_row->dtype);
  picked_col = picked_col.CreateView({new_len}, picked_col->dtype);
  picked_idx = picked_idx.CreateView({new_len}, picked_idx->dtype);

  hipDeviceSynchronize();
  return COOMatrix(
      mat.num_rows, mat.num_cols, picked_row, picked_col, picked_idx);
}

template <DGLDeviceType XPU, typename IdType>
COOMatrix CSRRowWiseSamplingUniform(
    CSRMatrix mat, IdArray rows, const int64_t num_picks, const bool replace) {
  if (num_picks == -1) {
    // Basically this is UnitGraph::InEdges().
    COOMatrix coo = CSRToCOO(CSRSliceRows(mat, rows), false);
    IdArray sliced_rows = IndexSelect(rows, coo.row);
    return COOMatrix(
        mat.num_rows, mat.num_cols, sliced_rows, coo.col, coo.data);
  } else {
    return _CSRRowWiseSamplingUniform<XPU, IdType>(
        mat, rows, num_picks, replace);
  }
}

uint historical_max_queue_size = 0;
std::vector<COOMatrix> CustomCSRRowWiseSamplingUniformTaskParallelism(
        CSRMatrix mat, IdArray rows, const IdArray &num_picks) {
//    std::printf("CustomCSRRowWiseSamplingUniformTaskParallelism run here\n");
    const auto& ctx = rows->ctx;
    const auto& num_picks_vec = num_picks.ToVector<int64_t>();
    auto device = runtime::DeviceAPI::Get(ctx);
    hipStream_t stream = runtime::getCurrentCUDAStream();

    // 1-hop seed nodes number
    const int64_t num_rows = rows->shape[0];
    const auto hops = num_picks->shape[0];

    // rows(i.e. batch nodes id) static cast to int64_t array, also a device ptr
    const int64_t* const sliced_rows = static_cast<const int64_t*>(rows->data);

    // for COO result
    // picked_row, picked_col, picked_idx: IdArray(NdArray)
    // out_row, out_col, out_idx: The data pointer points to the allocated data space(here is device ptr)
    std::vector<IdArray> picked_rows(hops), picked_cols(hops), picked_indices(hops);
    const int64_t* in_ptr = static_cast<int64_t*>(GetDevicePointer(mat.indptr));
    const int64_t* in_cols = static_cast<int64_t*>(GetDevicePointer(mat.indices));
    const int64_t* data = CSRHasData(mat)
                          ? static_cast<int64_t*>(GetDevicePointer(mat.data))
                          : nullptr;
    const int64_t* num_picks_ptr = static_cast<int64_t*>(GetDevicePointer(num_picks));

    // allocate space for stdgpu container
    uint queue_cap = num_rows;
    // last hop sample result do not need to enqueue
    for (int i = 0; i < hops - 1; i++)
        queue_cap += queue_cap * (num_picks_vec[i] + 1);

    nvtxRangePushA("create task_queue");
    auto task_queue = static_cast<thrust::pair<short, int64_t> *>(device->AllocWorkspace(ctx, queue_cap * sizeof(thrust::pair<short, int64_t>)));
//    auto task_queue = stdgpu::queue<thrust::pair<short, int64_t>>::createDeviceObject(queue_cap);
    nvtxRangePop();

    nvtxRangePushA("create bits");
    const auto bits_size = mat.num_rows * (hops - 1);
    uint* bool_arr = static_cast<uint *>(device->AllocWorkspace(ctx, bits_size * sizeof(uint)));
    //TODO: reset in init_kernel kernel(maybe better)
    CUDA_CALL(hipMemset(bool_arr, 0, bits_size * sizeof(uint)));
    nvtxRangePop();

    // init
    const dim3 init_block(512);
//    const dim3 init_grid((num_rows + init_block.x - 1) / init_block.x);
//    const dim3 init_grid((num_rows * hops + init_block.x - 1) / init_block.x);
    const dim3 init_grid((queue_cap + init_block.x - 1) / init_block.x);
    CUDA_KERNEL_CALL((init_kernel), init_grid, init_block, 0, stream, task_queue, bool_arr, sliced_rows, num_rows, hops, queue_cap);
//    assert(task_queue.size() == num_rows);

    nvtxRangePushA("create array of structs of array");
    // use SoA instead of AoS, which do not need to use thrust::copy_if
    selectedEdgeInfo* struct_arr_h = (selectedEdgeInfo*)malloc(hops * sizeof(selectedEdgeInfo));
    int64_t node_num = num_rows;
    for(int i = 0; i < hops; i++) {
        int64_t hop_res_num = node_num * num_picks_vec[i];
        picked_rows[i] = NewIdArray(hop_res_num, ctx, sizeof(int64_t) * 8);
        picked_cols[i] = NewIdArray(hop_res_num, ctx, sizeof(int64_t) * 8);
        picked_indices[i] = NewIdArray(hop_res_num, ctx, sizeof(int64_t) * 8);
        struct_arr_h[i].rows = static_cast<int64_t*>(picked_rows[i]->data);
        struct_arr_h[i].cols = static_cast<int64_t*>(picked_cols[i]->data);
        struct_arr_h[i].datas = static_cast<int64_t*>(picked_indices[i]->data);
        node_num += hop_res_num;
    }
    //TODO: res_vector can be reused, and need not to reset everytime because we "push" the result by atomic index!
    // and then using `CreateView` to get the corrct result
    auto struct_arr_d = static_cast<selectedEdgeInfo *>(device->AllocWorkspace(ctx, hops * sizeof(selectedEdgeInfo)));
    CUDA_CALL(hipMemcpy(struct_arr_d, struct_arr_h, sizeof(selectedEdgeInfo) * hops, hipMemcpyHostToDevice));
    nvtxRangePop();

//    const uint64_t random_seed = RandomEngine::ThreadLocal()->RandInt(1000000000);
    // fix only for reproduce!
    const uint64_t random_seed = 1234;

    const dim3 block(BLOCK_SIZE_CUSTOM);
//    const dim3 grid(num_rows);
    uint est_queue_cap;
    if (historical_max_queue_size == 0)
        est_queue_cap = queue_cap;
    else
        // more extreme
        est_queue_cap = historical_max_queue_size + num_rows;
    const dim3 grid(est_queue_cap);
//    const dim3 grid(num_rows * hops);
//    const dim3 grid(1);

    uint* vector_lens = static_cast<uint *>(device->AllocWorkspace(ctx, hops * sizeof(uint)));
    CUDA_CALL(hipMemset(vector_lens, 0, hops * sizeof(uint)));
    uint* vector_lens_h = (uint *) malloc(hops * sizeof(uint));
//    std::printf("cuda kernel launched\n");
    CUDA_KERNEL_CALL((_CSRRowWiseSampleUniformTaskParallelismKernel), grid, block, 0, stream,
                     random_seed, num_picks_ptr, vector_lens, num_rows, hops, mat.num_rows, in_ptr, in_cols, data,
                     task_queue, bool_arr, struct_arr_d);
//    assert(task_queue.empty());
//    CUDA_CALL(hipDeviceSynchronize());

//    std::printf("cuda kernel finished\n");

    // 传多个COO res的row, col, idx的指针的指针，用res_vector取fill，逻辑上最直观. 传指针的指针要写个demo试一下
    nvtxRangePushA("get result");
    //TODO: cannot overlap with host(DGL can)
    CUDA_CALL(hipMemcpy(vector_lens_h, vector_lens, sizeof(uint) * hops, hipMemcpyDeviceToHost));
    std::vector<COOMatrix> ret_coo(hops);
    for(int i = 0; i < hops; i++) {
        picked_rows[i] = picked_rows[i].CreateView({vector_lens_h[i]}, picked_rows[i]->dtype);
        picked_cols[i] = picked_cols[i].CreateView({vector_lens_h[i]}, picked_cols[i]->dtype);
        picked_indices[i] = picked_indices[i].CreateView({vector_lens_h[i]}, picked_indices[i]->dtype);
        ret_coo[i] = COOMatrix(mat.num_rows, mat.num_cols, picked_rows[i], picked_cols[i], picked_indices[i]);
    }
    nvtxRangePop();

    //TODO: we can do this only one epoch
    uint actual_queue_size;
    CUDA_CALL(hipMemcpyFromSymbol(&actual_queue_size, HIP_SYMBOL(tail_index), sizeof(uint), 0));
    // for correctness
    // (if not all producer has finished their job, actual_queue_size is not the corrct value too)
    assert(actual_queue_size <= est_queue_cap);
    historical_max_queue_size = std::max(historical_max_queue_size, actual_queue_size);

    nvtxRangePushA("free container");
    free(struct_arr_h);
    free(vector_lens_h);
    device->FreeWorkspace(ctx, bool_arr);
    device->FreeWorkspace(ctx, struct_arr_d);
    device->FreeWorkspace(ctx, vector_lens);
    device->FreeWorkspace(ctx, task_queue);
    nvtxRangePop();

//    std::printf("CustomCSRRowWiseSamplingUniformTaskParallelism finished here\n");
    return ret_coo;
}

template COOMatrix CSRRowWiseSamplingUniform<kDGLCUDA, int32_t>(
    CSRMatrix, IdArray, int64_t, bool);
template COOMatrix CSRRowWiseSamplingUniform<kDGLCUDA, int64_t>(
    CSRMatrix, IdArray, int64_t, bool);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
